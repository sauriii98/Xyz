
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 10

__global__ void VectorMatrixMult(int a[], int b[], int c[], int n)
{
	int i = threadIdx.x;

	if(i < n)
	{
		for(int j=0; j<SIZE; j++)
		{
			c[i] = c[i] + (a[j] * *(b + i*SIZE + j));
		}
	}
}

int main()
{
	int *a, *b, *c;
	
	a = (int*)malloc(SIZE * sizeof(int));
	b = (int*)malloc(SIZE * SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));
	
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = i+1;
		for (int j = 0; j < SIZE; j++)
		{
			*(b + i*SIZE + j) = i*j;
		}
	}
	
	int *d_a, *d_b, *d_c;
	
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));
	
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
	
	VectorMatrixMult <<< 1, SIZE >>> (d_a, d_b, d_c, SIZE);
	
	hipDeviceSynchronize();
	
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	printf("Vector: \n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", a[i]);
	}
	printf("\n");
	printf("Matrix: \n");
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(b + i*SIZE + j));
		}
		printf("\n");
	}
	printf("Product: \n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", c[i]);
	}
	printf("\n");
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	free(a);
	free(b);
	free(c);

	return 0;
}
