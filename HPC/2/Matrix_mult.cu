#include "hip/hip_runtime.h"
#include<stdio.h>
#define SIZE 2

__global__ void MatrixMul(int a[], int b[], int c[], int n) 
{ int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int sum = 0;
    for(int j = 0; j < n; j++) {
        sum += a[row * n + j] * b[j * n + col];
    }
    c[n*row + col] = sum;
}
int main() 
{    
	time_t t;
	srand((unsigned) time(&t));
	
	int *a, *b, *c;
	
	a = (int*)malloc(SIZE * SIZE * sizeof(int));
	b = (int*)malloc(SIZE * SIZE * sizeof(int));
	c = (int*)malloc(SIZE * SIZE * sizeof(int));
	
    	for(int i = 0; i < SIZE ; i++) 
    	{
        	for(int j = 0; j < SIZE ; j++) 
        	{
        		*(a + i*SIZE + j) = i;
        		*(b + i*SIZE + j) = i+1;
        	}
    	}
    
	int *d_a, *d_b, *d_c;

    	hipMalloc(&d_a, SIZE * SIZE * sizeof(int));
    	hipMalloc(&d_b, SIZE * SIZE * sizeof(int));
    	hipMalloc(&d_c, SIZE * SIZE * sizeof(int));
    	
    	hipMemcpy(d_a, a, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(d_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 grid_dim(SIZE, SIZE, 1);
    MatrixMul<<<grid_dim, 1>>> (d_a, d_b, d_c, SIZE);
    
    	
    	hipDeviceSynchronize();

    	hipMemcpy(c, d_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);
    
    	printf("1st matrix: ");
	for (int i = 0; i < SIZE; i++)
	{
		for(int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(a + i*SIZE + j));
		}
		printf("");
	}
	printf("2nd matrix: ");
	for (int i = 0; i < SIZE; i++)
	{
		for(int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(b + i*SIZE + j));
		}
		printf("");
	}
	printf("Product: ");
	for (int i = 0; i < SIZE; i++)
	{
		for(int j = 0; j < SIZE; j++)
		{
			printf("%d ", *(c + i*SIZE + j));
		}
		printf("");
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	free(a);
	free(b);
	free(c);

   
    	return 0;
}

