#include "hip/hip_runtime.h"
#include <iostream>
usingb namespace std;

__global__ void addi(int *a,int *b,int *c,int n)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if(tid<n)
		c[tid] = a[tid] + b[tid];
}

int main()
{
	int *a,*b,*c;
	int n;
	int *d_a,*d_b,*d_c;	

	cin>>n;

	int size = n*sizeof(int);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for(int i=0;i<n;i++)
	{
		a[i] = i;
		b[i] = i+1;
		c[i] = 0;
	}

	hipMalloc(&d_a,size);
	hipMalloc(&d_b,size);
	hipMalloc(&d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);

	addi<<<2,n/2>>>(d_a,d_b,d_c,n);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
	{
		cout<<a[i]<<" + "<<b[i]<<"  = "<<c[i]<<endl; 
	}

	return 0;
}