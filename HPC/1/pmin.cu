
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void mini(int *a)
{
	int tid = threadIdx.x;
	int step_size = 1;
	int n_thread = blockDim.x;
	int f,s;

	while(n_thread>0)
	{
		if(tid<n_thread)
		{
			f = tid*step_size*2;
			s = f + step_size;
			if(a[f]>=a[s])
				a[f] = a[s];
		}
		step_size<<=1;
		n_thread>>=1;
	}
}

int main()
{
	int *a,*b,size,n;
	int *d_a;

	cin>>n;

	size = n*sizeof(int);

	a = (int *)malloc(size);
	b = (int *)malloc(sizeof(int));

	hipMalloc(&d_a,size);

	for(int i=0;i<n;i++)
	{
		a[i] = rand()%100;
	}

	for(int i=0;i<n;i++)
	{
		cout<<a[i]<<" ";
	}

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

		clock_t start = clock();


	mini<<<1,n/2>>>(d_a);

	cout<<"time: "<<(float)(clock()-start)/CLOCKS_PER_SEC<<endl;

	hipMemcpy(b,d_a,size,hipMemcpyDeviceToHost);

	cout<<"min is :"<<b[0];

	return 0;
}