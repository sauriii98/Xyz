
#include <hip/hip_runtime.h>
#include <iostream>
#include<ctime>
using namespace std;

__global__ void mini(int *a,int *b,int n)
{
	int tid = threadIdx.x;
	int minn = INT_MAX;
	for(int i=0;i<min(tid+256,n);i++)
	{
		if(minn>a[i])
			minn = a[i];
	}
	b[tid] = minn;
}

int main()
{
	int *a,*b,size,n;
	int *d_a,*d_b;

	cin>>n;
	size = n*sizeof(int);

	a = (int*)malloc(size);
	b = (int *)malloc(sizeof(int));

	hipMalloc(&d_a,size);
	hipMalloc(&d_b,sizeof(int));

	for(int i=0;i<n;i++)
	{
		a[i] = rand()%100;
	}

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	clock_t start = clock();

	mini<<<1,n>>>(d_a,d_b,n);

	cout<<"time: "<<(float)(clock()-start)/CLOCKS_PER_SEC;

	hipMemcpy(b,d_b,sizeof(int),hipMemcpyDeviceToHost);

	cout<<"min is:"<<b[0];

	return 0;
}